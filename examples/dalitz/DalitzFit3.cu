// ROOT stuff
#include "TRandom.hh"
#include "TCanvas.h" 
#include "TFile.h" 
#include "TH1F.h" 
#include "TH2F.h" 
#include "TStyle.h" 
#include "TRandom3.hh" 
#include "TLegend.h" 
#include "TText.h" 
#include "TLine.h" 

// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

// GooFit stuff
#include "Variable.hh" 
#include "PolynomialPdf.hh" 
#include "DalitzPlotPdf.hh" 
#include "DalitzVetoPdf.hh" 
#include "ResonancePdf.hh" 
#include "AddPdf.hh"
#include "ProdPdf.hh"
#include "GooPdf.hh" 
#include "FitManager.hh" 
#include "UnbinnedDataSet.hh"
#include "EventWeightedAddPdf.hh"
#include "SmoothHistogramPdf.hh"

using namespace std;

TCanvas* foo; 
TCanvas* foodal; 
timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 
UnbinnedDataSet* data = 0; 
TH2F* weightHistogram = 0; 
TH2F* underlyingBins = 0; 

Variable* m12 = 0;
Variable* m13 = 0;
Variable* eventNumber = 0; 
Variable* wSig0 = 0; 
bool fitMasses = false; 
Variable* fixedRhoMass  = new Variable("rho_mass", 0.7758, 0.01, 0.7, 0.8);
Variable* fixedRhoWidth = new Variable("rho_width", 0.1503, 0.01, 0.1, 0.2); 

const fptype _mD0 = 1.86484; 
const fptype _mD02 = _mD0 *_mD0;
const fptype _mD02inv = 1./_mD02; 
const fptype piPlusMass = 0.13957018;
const fptype piZeroMass = 0.1349766;

// Constants used in more than one PDF component. 
Variable* motherM = new Variable("motherM", _mD0);
Variable* chargeM = new Variable("chargeM", piPlusMass);
Variable* neutrlM = new Variable("neutrlM", piZeroMass);
Variable* massSum = new Variable("massSum", _mD0*_mD0 + 2*piPlusMass*piPlusMass + piZeroMass*piZeroMass); // = 3.53481 
Variable* veto_min = new Variable("veto_min", 0.475*0.475);
Variable* veto_max = new Variable("veto_min", 0.505*0.505);
Variable* constantOne = new Variable("constantOne", 1); 
Variable* constantZero = new Variable("constantZero", 0); 
  
std::vector<PdfBase*> comps;

GooPdf* kzero_veto = 0; 
char strbuffer[1000]; 
double mesonRad  = 1.5;
DalitzPlotPdf* signalDalitz; 
bool doEffSwap = false;
bool saveEffPlot = true;

fptype cpuGetM23 (fptype massPZ, fptype massPM) {
  return (_mD02 + piZeroMass*piZeroMass + piPlusMass*piPlusMass + piPlusMass*piPlusMass - massPZ - massPM); 
}

bool cpuDalitz (fptype m12, fptype m13, fptype bigM, fptype dm1, fptype dm2, fptype dm3) {
  if (m12 < POW(dm1 + dm2, 2)) return false; // This m12 cannot exist, it's less than the square of the (1,2) particle mass.
  if (m12 > POW(bigM - dm3, 2)) return false;   // This doesn't work either, there's no room for an at-rest 3 daughter. 
  
  // Calculate energies of 1 and 3 particles in m12 rest frame. 
  fptype e1star = 0.5 * (m12 - dm2*dm2 + dm1*dm1) / SQRT(m12); 
  fptype e3star = 0.5 * (bigM*bigM - m12 - dm3*dm3) / SQRT(m12); 

  // Bounds for m13 at this value of m12.
  fptype minimum = POW(e1star + e3star, 2) - POW(SQRT(e1star*e1star - dm1*dm1) + SQRT(e3star*e3star - dm3*dm3), 2);
  if (m13 < minimum) return false;
  fptype maximum = POW(e1star + e3star, 2) - POW(SQRT(e1star*e1star - dm1*dm1) - SQRT(e3star*e3star - dm3*dm3), 2);
  if (m13 > maximum) return false;

  return true; 
}

void getToyData (std::string toyFileName, double sigweight = 0.9) {
  TH2F dalitzplot("dalitzplot", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit); 
  std::vector<Variable*> vars;
  vars.push_back(m12);
  vars.push_back(m13);
  vars.push_back(eventNumber); 
  vars.push_back(wSig0); 
  data = new UnbinnedDataSet(vars); 

  std::ifstream reader;
  reader.open(toyFileName.c_str()); 
  std::string buffer;
  while (!reader.eof()) {
    reader >> buffer;
    if (buffer == "====") break; 
    std::cout << buffer; 
  }

  double dummy = 0; 
  wSig0->value = sigweight;
  while (!reader.eof()) {
    reader >> dummy;
    reader >> dummy;      // m23, m(pi+ pi-), called m12 in processToyRoot convention. 
    reader >> m12->value; // Already swapped according to D* charge. m12 = m(pi+pi0)
    reader >> m13->value;

    // Errors on Dalitz variables
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 

    reader >> dummy; // Decay time
    reader >> dummy; // sigma_t

    reader >> dummy; // Md0
    reader >> dummy; // deltaM
    reader >> dummy; // ProbSig
    reader >> dummy; // Dst charge
    reader >> dummy; // Run
    reader >> dummy; // Event
    reader >> dummy; // Signal and four bkg fractions. 
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 

    // EXERCISE 1 (preliminary): Impose an artificial reconstruction efficiency
    // by throwing out events with a probability linear in m12. 
    // NB! This exercise continues below. 

    // EXERCISE 2: Instead of the above efficiency, impose a 
    // K0 veto, by throwing out events with 0.475 < m23 < 0.505. 
    double M_23 = cpuGetM23(m12->value, m13->value);
    if (M_23>veto_min->value && M_23<veto_max->value) continue;

    // EXERCISE 3: Use both the above. 

    eventNumber->value = data->getNumEvents(); 
    data->addEvent(); 

    dalitzplot.Fill(m12->value, m13->value); 
  }
  reader.close();
  
  TRandom3 donram(0); 
  int nsig = data->getNumEvents();
  // Generate background events based on a flat distribution across DP plane
  for (int ib = 0; ib < nsig*(1-sigweight)/sigweight; ib++){
    do{
    m12->value = donram.Uniform(m12->lowerlimit, m12->upperlimit);
    m13->value = donram.Uniform(m13->lowerlimit, m13->upperlimit);
    }while(!cpuDalitz(m12->value, m13->value, _mD0, piZeroMass, piPlusMass, piPlusMass));
    double M_23 = cpuGetM23(m12->value, m13->value);
    if (M_23>veto_min->value && M_23<veto_max->value) {ib--; continue;}
    eventNumber->value = data->getNumEvents(); 
    data->addEvent(); 
    dalitzplot.Fill(m12->value, m13->value); 
  }

  dalitzplot.SetStats(false); 
  dalitzplot.Draw("colz");
  foodal->SaveAs("dalitzplot.png"); 
}

//Taking care of bin blocks around the DP boundary
void createWeightHistogram () {
  weightHistogram = new TH2F("weightHistogram", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit);
  weightHistogram->SetStats(false); 
  double step12 = (m12->upperlimit - m12->lowerlimit) / m12->numbins;
  double step13 = (m13->upperlimit - m13->lowerlimit) / m13->numbins;

  for (int i = 1; i <= m12->numbins; ++i) {
    for (int j = 1; j < m13->numbins; ++j) {
      double maxCount = 0;
      double count = 0; 
      for (double currM12 = m12->lowerlimit + step12*(i-1) + 0.05*step12; currM12 < m12->lowerlimit + step12*i; currM12 += 0.1*step12) {
	for (double currM13 = m13->lowerlimit + step13*(j-1) + 0.05*step13; currM13 < m13->lowerlimit + step13*j; currM13 += 0.1*step13) {
	  maxCount++;
	  if (!cpuDalitz(currM12, currM13, _mD0, piZeroMass, piPlusMass, piPlusMass)) continue;
	  count++; 
	}
      }
      if (0.1 > maxCount) continue;
      count /= maxCount;
      weightHistogram->SetBinContent(i, j, count); 
    }
  }
  //Histogram used to draw eff. plot
  underlyingBins = new TH2F("underlyingBins", "", 
			    m12->numbins, m12->lowerlimit, m12->upperlimit, 
			    m13->numbins, m13->lowerlimit, m13->upperlimit);
  underlyingBins->SetStats(false); 
}

GooPdf* makeEfficiencyPdf () {
  vector<Variable*> lvars;
  lvars.push_back(m12); 
  lvars.push_back(m13);  
  BinnedDataSet* binEffData = new BinnedDataSet(lvars); 
  createWeightHistogram();
  // Now testing your efficiency data by uniformly generating m12,m13 values 
  TRandom3 donram(0); 
  for (int i = 0; i < 1e5; i++){
    do{
    m12->value = donram.Uniform(m12->lowerlimit, m12->upperlimit);
    m13->value = donram.Uniform(m13->lowerlimit, m13->upperlimit);
    }while(!cpuDalitz(m12->value, m13->value, _mD0, piZeroMass, piPlusMass, piPlusMass));
    //Weight will not be one if the physics boundary crosses the bin square.
    double weight = weightHistogram->GetBinContent(weightHistogram->FindBin(m12->value, m13->value));
    binEffData->addWeightedEvent(weight);
    if (underlyingBins) underlyingBins->Fill(m12->value, m13->value, weight);
    // Imposing the requirement on efficiency symmetry along m12=m13 line
      if (doEffSwap){
      double swapmass = m12->value; m12->value = m13->value; m13->value = swapmass;
      weight = weightHistogram->GetBinContent(weightHistogram->FindBin(m12->value, m13->value));
      binEffData->addWeightedEvent(weight);
      if (underlyingBins) underlyingBins->Fill(m12->value, m13->value, weight);
      swapmass = m12->value; m12->value = m13->value; m13->value = swapmass;   
      }
  }
  if (saveEffPlot) {
    foodal->cd();
    underlyingBins->Draw("colz"); 
    foodal->SaveAs("plots/efficiency_bins.png");
    foodal->SetLogz(true);
    foodal->SaveAs("plots/efficiency_bins_log.png");
    foo->cd(); 
  }

  //Variable* effSmoothing = new Variable("effSmoothing", 1.0, 0.1, 0, 1.25); 
  Variable* effSmoothing = new Variable("effSmoothing", 0);   
  SmoothHistogramPdf* ret = new SmoothHistogramPdf("efficiency", binEffData, effSmoothing); 

  return ret; 
}

GooPdf* makeKzeroVeto () {
  if (kzero_veto) return kzero_veto; 

  VetoInfo* kVetoInfo = new VetoInfo();
  kVetoInfo->cyclic_index = PAIR_23; 
  kVetoInfo->minimum = veto_min;
  kVetoInfo->maximum = veto_max;
  vector<VetoInfo*> vetos; vetos.push_back(kVetoInfo); 
  kzero_veto = new DalitzVetoPdf("kzero_veto", m12, m13, motherM, neutrlM, chargeM, chargeM, vetos); 
  return kzero_veto;
}

GooPdf* makeFlatBkgDalitzPdf() {
  VetoInfo* kVetoInfo = new VetoInfo();
  kVetoInfo->cyclic_index = PAIR_23; 
  kVetoInfo->minimum = veto_min;
  kVetoInfo->maximum = veto_max;
  vector<VetoInfo*> vetos; vetos.push_back(kVetoInfo); 
  GooPdf *ret = new DalitzVetoPdf("flatbkgPdf", m12, m13, motherM, neutrlM, chargeM, chargeM, vetos); 
  return ret;
}

DalitzPlotPdf* makeSignalPdf (GooPdf* eff = 0) {
  DecayInfo* dtop0pp = new DecayInfo();
  dtop0pp->motherMass  = _mD0; 
  dtop0pp->daug1Mass  = piZeroMass;
  dtop0pp->daug2Mass  = piPlusMass;
  dtop0pp->daug3Mass  = piPlusMass;
  dtop0pp->meson_radius  = 1.5; 
 
  ResonancePdf* rhop  = new ResonancePdf("rhop",
							     new Variable("rhop_amp_real", 1),
							     new Variable("rhop_amp_imag", 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_12);


  bool fixAmps = false;

  ResonancePdf* rhom  = new ResonancePdf("rhom", 
							     fixAmps ? new Variable("rhom_amp_real", 0.714) : 
							     new Variable("rhom_amp_real",  0.714, 0.001, 0, 0),
							     fixAmps ? new Variable("rhom_amp_imag", -0.025) :
							     new Variable("rhom_amp_imag", -0.025, 0.1, 0, 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_13);

  ResonancePdf* rho0  = new ResonancePdf("rho0", 
							     fixAmps ? new Variable("rho0_amp_real", 0.565) : 
							     new Variable("rho0_amp_real", 0.565, 0.001, 0, 0),
							     fixAmps ? new Variable("rho0_amp_imag", 0.164) :
							     new Variable("rho0_amp_imag", 0.164, 0.1, 0, 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_23);

  Variable* sharedMass = new Variable("rhop_1450_mass", 1.465, 0.01, 1.0, 2.0);
  Variable* shareWidth = new Variable("rhop_1450_width", 0.400, 0.01, 0.01, 5.0); 

  ResonancePdf* rhop_1450  = new ResonancePdf("rhop_1450", 
								  fixAmps ? new Variable("rhop_1450_amp_real", -0.174) : 
								  new Variable("rhop_1450_amp_real", -0.174, 0.001, 0, 0),
								  fixAmps ? new Variable("rhop_1450_amp_imag", -0.117) :
								  new Variable("rhop_1450_amp_imag", -0.117, 0.1, 0, 0),
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_12);

  ResonancePdf* rho0_1450  = new ResonancePdf("rho0_1450", 
								  fixAmps ? new Variable("rho0_1450_amp_real", 0.325) : 
								  new Variable("rho0_1450_amp_real", 0.325, 0.001, 0, 0),
								  fixAmps ? new Variable("rho0_1450_amp_imag", 0.057) : 
								  new Variable("rho0_1450_amp_imag", 0.057, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_23);

  ResonancePdf* rhom_1450  = new ResonancePdf("rhom_1450", 
								  fixAmps ? new Variable("rhom_1450_amp_real", 0.788) : 
								  new Variable("rhom_1450_amp_real", 0.788, 0.001, 0, 0),
								  fixAmps ? new Variable("rhom_1450_amp_imag", 0.226) : 
								  new Variable("rhom_1450_amp_imag", 0.226, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_13);

  sharedMass = new Variable("rhop_1700_mass",  1.720, 0.01, 1.6, 1.9);
  shareWidth = new Variable("rhop_1700_width", 0.250, 0.01, 0.1, 1.0); 

  
  ResonancePdf* rhop_1700  = new ResonancePdf("rhop_1700", 
								  fixAmps ? new Variable("rhop_1700_amp_real", 2.151) : 
								  new Variable("rhop_1700_amp_real",  2.151, 0.001, 0, 0),
								  fixAmps ? new Variable("rhop_1700_amp_imag", -0.658) : 
								  new Variable("rhop_1700_amp_imag", -0.658, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_12);
  
  ResonancePdf* rho0_1700  = new ResonancePdf("rho0_1700", 
								  fixAmps ? new Variable("rho0_1700_amp_real",  2.400) : 
								  new Variable("rho0_1700_amp_real",  2.400, 0.001, 0, 0),
								  fixAmps ? new Variable("rho0_1700_amp_imag", -0.734) : 
								  new Variable("rho0_1700_amp_imag", -0.734, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_23);
  
  ResonancePdf* rhom_1700  = new ResonancePdf("rhom_1700", 
								  fixAmps ? new Variable("rhom_1700_amp_real",  1.286) : 
								  new Variable("rhom_1700_amp_real",  1.286, 0.001, 0, 0),
								  fixAmps ? new Variable("rhom_1700_amp_imag", -1.532) : 
								  new Variable("rhom_1700_amp_imag", -1.532, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_13);
  
  ResonancePdf* f0_980  = new ResonancePdf("f0_980", 
							       fixAmps ? new Variable("f0_980_amp_real",  0.008 * (-_mD02)) : 
							       new Variable("f0_980_amp_real",  0.008 * (-_mD02), 0.001, 0, 0),
							       fixAmps ? new Variable("f0_980_amp_imag", -0.013 * (-_mD02)) : 
							       new Variable("f0_980_amp_imag", -0.013 * (-_mD02), 0.1, 0, 0),  
							       new Variable("f0_980_mass",     0.980, 0.01, 0.8, 1.2),
							       new Variable("f0_980_width",    0.044, 0.001, 0.001, 0.08),
							       (unsigned int)0,
							       PAIR_23);
  
  ResonancePdf* f0_1370  = new ResonancePdf("f0_1370", 
								fixAmps ? new Variable("f0_1370_amp_real", -0.058 * (-_mD02)) : 
								new Variable("f0_1370_amp_real", -0.058 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1370_amp_imag",  0.026 * (-_mD02)) : 
								new Variable("f0_1370_amp_imag",  0.026 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1370_mass",     1.434, 0.01, 1.2, 1.6),
								new Variable("f0_1370_width",    0.173, 0.01, 0.01, 0.4),
							    (unsigned int)0,
								PAIR_23);
  
  ResonancePdf* f0_1500  = new ResonancePdf("f0_1500", 
								fixAmps ? new Variable("f0_1500_amp_real", 0.057 * (-_mD02)) : 
								new Variable("f0_1500_amp_real", 0.057 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1500_amp_imag", 0.012 * (-_mD02)) : 
								new Variable("f0_1500_amp_imag", 0.012 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1500_mass",     1.507, 0.01, 1.3, 1.7),
								new Variable("f0_1500_width",    0.109, 0.01, 0.01, 0.3),
							    (unsigned int)0,
								PAIR_23);
  
  ResonancePdf* f0_1710  = new ResonancePdf("f0_1710", 
								fixAmps ? new Variable("f0_1710_amp_real", 0.070 * (-_mD02)) : 
								new Variable("f0_1710_amp_real", 0.070 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1710_amp_imag", 0.087 * (-_mD02)) : 
								new Variable("f0_1710_amp_imag", 0.087 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1710_mass",     1.714, 0.01, 1.5, 2.9), 
								new Variable("f0_1710_width",    0.140, 0.01, 0.01, 0.5),
								(unsigned int)0,
								PAIR_23);
  
  ResonancePdf* f2_1270  = new ResonancePdf("f2_1270", 
								fixAmps ? new Variable("f2_1270_amp_real", -1.027 * (-_mD02inv)) : 
								new Variable("f2_1270_amp_real", -1.027 * (-_mD02inv), 0.001, 0, 0),
								fixAmps ? new Variable("f2_1270_amp_imag", -0.162 * (-_mD02inv)) : 
								new Variable("f2_1270_amp_imag", -0.162 * (-_mD02inv), 0.1, 0, 0),  
								new Variable("f2_1270_mass",     1.2754, 0.01, 1.0, 1.5),
								new Variable("f2_1270_width",    0.1851, 0.01, 0.01, 0.4),
								2,
								PAIR_23);
  
  ResonancePdf* f0_600  = new ResonancePdf("f0_600", 
							       fixAmps ? new Variable("f0_600_amp_real", 0.068 * (-_mD02)) : 
							       new Variable("f0_600_amp_real", 0.068 * (-_mD02), 0.001, 0, 0),
							       fixAmps ? new Variable("f0_600_amp_imag", 0.010 * (-_mD02)) : 
							       new Variable("f0_600_amp_imag", 0.010 * (-_mD02), 0.1, 0, 0),  
							       new Variable("f0_600_mass",     0.500, 0.01, 0.3, 0.7),
							       new Variable("f0_600_width",    0.400, 0.01, 0.2, 0.6), 
							       (unsigned int)0,
							       PAIR_23);
  
  ResonancePdf* nonr  = new ResonancePdf("nonr",
							     fixAmps ? new Variable("nonr_amp_real", 0.5595 * (-1)) : 
							     new Variable("nonr_amp_real", 0.5595 * (-1),   0.001, 0, 0),
							     fixAmps ? new Variable("nonr_amp_imag", -0.108761 * (-1)) : 
							     new Variable("nonr_amp_imag", -0.108761* (-1), 0.1, 0, 0)); 

  dtop0pp->resonances.push_back(nonr); 
  dtop0pp->resonances.push_back(rhop);
  dtop0pp->resonances.push_back(rho0); 
  dtop0pp->resonances.push_back(rhom); 
  dtop0pp->resonances.push_back(rhop_1450); 
  dtop0pp->resonances.push_back(rho0_1450); 
  dtop0pp->resonances.push_back(rhom_1450); 
  dtop0pp->resonances.push_back(rhop_1700); 
  dtop0pp->resonances.push_back(rho0_1700); 
  dtop0pp->resonances.push_back(rhom_1700); 
  dtop0pp->resonances.push_back(f0_980); 
  dtop0pp->resonances.push_back(f0_1370); 
  dtop0pp->resonances.push_back(f0_1500); 
  dtop0pp->resonances.push_back(f0_1710); 
  dtop0pp->resonances.push_back(f2_1270); 
  dtop0pp->resonances.push_back(f0_600); 

  if (!fitMasses) {
    for (vector<ResonancePdf*>::iterator res = dtop0pp->resonances.begin(); res != dtop0pp->resonances.end(); ++res) {
      (*res)->setParameterConstantness(true); 
    }
  }

  if (!eff) {
    // By default create a constant efficiency. 
    vector<Variable*> offsets;
    vector<Variable*> observables;
    vector<Variable*> coefficients; 

    observables.push_back(m12);
    observables.push_back(m13);
    offsets.push_back(constantZero);
    offsets.push_back(constantZero);
    coefficients.push_back(constantOne); 
    eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
  }
  comps.clear();
  comps.push_back(eff);
  if (!kzero_veto) makeKzeroVeto();
  comps.push_back(kzero_veto);
  ProdPdf* effWithVeto = new ProdPdf("effWithVeto", comps);

  return new DalitzPlotPdf("signalPDF", m12, m13, eventNumber, dtop0pp, effWithVeto);
}

void drawFitPlotsWithPulls(TH1* hd, TH1* ht, TH1* hb, string plotdir){
    const char* hname = hd->GetName();
    char obsname[10];
    for (int i=0;;i++) {
        if (hname[i]=='_') obsname[i] = '\0';
        else obsname[i] = hname[i];
        if (obsname[i] == '\0') break;
    }
    ht->Scale(hd->Integral()/ht->Integral());
    hb->Scale(hd->Integral()/ht->Integral());
    foo->cd(); 
    foo->Clear();
    hd->Draw("ep");
    ht->Draw("lsame");
    hb->SetLineStyle(kDashed);
    hb->Draw("lsame");
    sprintf(strbuffer, "%s/%s_fit.C", plotdir.c_str(), obsname);
    foo->SaveAs(strbuffer);
    sprintf(strbuffer, "%s/%s_fit.pdf", plotdir.c_str(), obsname);
    foo->SaveAs(strbuffer);
/*    sprintf(strbuffer, "%s/%s_fit_log.pdf", plotdir.c_str(), obsname);
    foo->SaveAs(strbuffer);*/
}

void makeToyDalitzPdfPlots (GooPdf* overallSignal, string plotdir = "plots") {
  TH1F m12_dat_hist("m12_dat_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit);
  m12_dat_hist.SetStats(false); 
  m12_dat_hist.SetMarkerStyle(8); 
  m12_dat_hist.SetMarkerSize(1.2);
  m12_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV]");
  sprintf(strbuffer, "Events / %.1f MeV", 1e3*m12_dat_hist.GetBinWidth(1));
  m12_dat_hist.GetYaxis()->SetTitle(strbuffer); 
  TH1F m12_pdf_hist("m12_pdf_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit);
  m12_pdf_hist.SetStats(false); 
  m12_pdf_hist.SetLineColor(kBlue); 
  m12_pdf_hist.SetLineWidth(3); 
  TH1* m12_pdf_hist_bkg = (TH1*)m12_pdf_hist.Clone("m12_pdf_hist_bkg");
  TH1F m13_dat_hist("m13_dat_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m13_dat_hist.SetStats(false); 
  m13_dat_hist.SetMarkerStyle(8); 
  m13_dat_hist.SetMarkerSize(1.2);
  m13_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV]");
  sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
  m13_dat_hist.GetYaxis()->SetTitle(strbuffer); 
  TH1F m13_pdf_hist("m13_pdf_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m13_pdf_hist.SetStats(false); 
  m13_pdf_hist.SetLineColor(kBlue); 
  m13_pdf_hist.SetLineWidth(3); 
  TH1* m13_pdf_hist_bkg = (TH1*)m13_pdf_hist.Clone("m13_pdf_hist_bkg");
  TH1F m23_dat_hist("m23_dat_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m23_dat_hist.SetStats(false); 
  m23_dat_hist.SetMarkerStyle(8); 
  m23_dat_hist.SetMarkerSize(1.2);
  m23_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{-}) [GeV]");
  sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
  m23_dat_hist.GetYaxis()->SetTitle(strbuffer); 
  TH1F m23_pdf_hist("m23_pdf_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m23_pdf_hist.SetStats(false); 
  m23_pdf_hist.SetLineColor(kBlue); 
  m23_pdf_hist.SetLineWidth(3); 
  TH1* m23_pdf_hist_bkg = (TH1*)m23_pdf_hist.Clone("m23_pdf_hist_bkg");
  TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit);
  dalitzpp0_dat_hist.SetStats(false); 
  dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV]");
  dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV]");
  TH2F dalitzpp0_pdf_hist("dalitzpp0_pdf_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit);
/*  dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(K^{-} #pi^{0}) [GeV^{2}]");
  dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(K^{-} #pi^{+}) [GeV^{2}]");*/
  dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV^{2}]");
  dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV^{2}]");
  dalitzpp0_pdf_hist.SetStats(false); 
  double totalPdf = 0; 
  double totalDat = 0; 
  double totalSigProb = 0;
  double totalBGProb = 0;
  int evtCounter = 0; 
  double totalPdf_bg = 0;
  for (unsigned int evt = 0; evt < data->getNumEvents(); ++evt) {
    double data_m12 = data->getValue(m12, evt);
    m12_dat_hist.Fill(data_m12); 
    double data_m13 = data->getValue(m13, evt);
    m13_dat_hist.Fill(data_m13); 
    dalitzpp0_dat_hist.Fill(data_m12, data_m13);
    m23_dat_hist.Fill(cpuGetM23(data_m12, data_m13)); 
    totalSigProb += data->getValue(wSig0, evt);    
    totalBGProb += 1 - data->getValue(wSig0, evt);    
    totalDat++; 
  }
  wSig0->value = totalSigProb / totalDat;
    std::vector<Variable*> vars;
    vars.push_back(m12);
    vars.push_back(m13);
    vars.push_back(eventNumber); 
    vars.push_back(wSig0);
    UnbinnedDataSet currData(vars); 

  for (int i = 0; i < m12->numbins; ++i) {
      m12->value = m12->lowerlimit + (m12->upperlimit - m12->lowerlimit)*(i + 0.5) / m12->numbins; 
      for (int j = 0; j < m13->numbins; ++j) {
          m13->value = m13->lowerlimit + (m13->upperlimit - m13->lowerlimit)*(j + 0.5) / m13->numbins; 
          if (!cpuDalitz(m12->value, m13->value, _mD0, piPlusMass, piPlusMass, piZeroMass)) continue;
          eventNumber->value = evtCounter; 
          evtCounter++;
          currData.addEvent(); 
      }
  }
  overallSignal->setData(&currData);
  signalDalitz->setDataSize(currData.getNumEvents(),4); 
  std::vector<std::vector<double> > pdfValues;
  overallSignal->getCompProbsAtDataPoints(pdfValues);
  for (unsigned int j = 0; j < pdfValues[0].size(); ++j) {
	double currm12 = currData.getValue(m12, j);
	double currm13 = currData.getValue(m13, j);

      dalitzpp0_pdf_hist.Fill(currm12, currm13, pdfValues[0][j]);
      m12_pdf_hist.Fill(currm12, pdfValues[0][j]);
      m13_pdf_hist.Fill(currm13, pdfValues[0][j]);
      m23_pdf_hist.Fill(cpuGetM23(currm12, currm13), pdfValues[0][j]); 
      // NB: 1st for signal, and 2nd for bkg.
      m12_pdf_hist_bkg->Fill(currm12, pdfValues[2][j]);
      m13_pdf_hist_bkg->Fill(currm13, pdfValues[2][j]);
      m23_pdf_hist_bkg->Fill(cpuGetM23(currm12, currm13), pdfValues[2][j]); 
      totalPdf     += pdfValues[0][j]; 
      totalPdf_bg     += pdfValues[2][j]; 
  }
/*  m12_pdf_hist_bkg->Scale(totalBGProb/totalPdf_bg);
  m13_pdf_hist_bkg->Scale(totalBGProb/totalPdf_bg);
  m23_pdf_hist_bkg->Scale(totalBGProb/totalPdf_bg);*/
  foodal->cd(); 
  foodal->SetLogz(false);
  dalitzpp0_pdf_hist.Draw("colz");
  foodal->SaveAs((plotdir + "/dalitzpp0_pdf.png").c_str());
/*  m12_pdf_hist.Draw("");
  foodal->SaveAs((plotdir + "/m12_pdf_hist.png").c_str());
  m13_pdf_hist.Draw("");
  foodal->SaveAs((plotdir + "/m13_pdf_hist.png").c_str());
  if (!data) return;*/
  dalitzpp0_dat_hist.Draw("colz");
  foodal->SaveAs((plotdir + "/dalitzpp0_dat.png").c_str());

  drawFitPlotsWithPulls(&m12_dat_hist, &m12_pdf_hist, m12_pdf_hist_bkg, plotdir);
  drawFitPlotsWithPulls(&m13_dat_hist, &m13_pdf_hist, m13_pdf_hist_bkg, plotdir);
  drawFitPlotsWithPulls(&m23_dat_hist, &m23_pdf_hist, m23_pdf_hist_bkg, plotdir);
  delete m12_pdf_hist_bkg;
  delete m13_pdf_hist_bkg;
  delete m23_pdf_hist_bkg;
}

void runToyFit (std::string toyFileName) {
  m12 = new Variable("m12", 0, 3);
  m13 = new Variable("m13", 0, 3); 
  m12->numbins = 240;
  m13->numbins = 240;
  eventNumber = new Variable("eventNumber", 0, INT_MAX);
  wSig0 = new Variable("wSig0", 0, 1);
  getToyData(toyFileName);

  // EXERCISE 1 (real part): Create a PolynomialPdf which models
  // the efficiency you imposed in the preliminary, and use it in constructing
  // the signal PDF. 

  // EXERCISE 2: Create a K0 veto function and use it as the efficiency. 

  // EXERCISE 3: Make the efficiency a product of the two functions
  // from the previous exercises.

  int oldBins1 = m12->numbins;
  int oldBins2 = m13->numbins;
  m12->numbins = 60;
  m13->numbins = 60;  //Use different choice of binning for efficiency
  GooPdf* eff = makeEfficiencyPdf();
  m12->numbins = oldBins1;
  m13->numbins = oldBins2;   
  signalDalitz = makeSignalPdf(eff); 
  signalDalitz->setDataSize(data->getNumEvents(),4); 
  GooPdf* bkgFlatPdf = makeFlatBkgDalitzPdf();
  std::vector<Variable*> evtWeights;
  evtWeights.push_back(wSig0);
  comps.clear();
  comps.push_back(signalDalitz);
  comps.push_back(bkgFlatPdf);
  EventWeightedAddPdf* totPdf = new EventWeightedAddPdf("total", evtWeights, comps);
  totPdf->setData(data); 
  FitManager datapdf(totPdf); 
  
  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);
  datapdf.fit(); 
  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  makeToyDalitzPdfPlots(totPdf);   
}

int main (int argc, char** argv) {
  gStyle->SetCanvasBorderMode(0);
  gStyle->SetCanvasColor(10);
  gStyle->SetFrameFillColor(10);
  gStyle->SetFrameBorderMode(0);
  gStyle->SetPadColor(0);
  gStyle->SetTitleColor(1);
  gStyle->SetStatColor(0);
  gStyle->SetFillColor(0);
  gStyle->SetFuncWidth(1);
  gStyle->SetLineWidth(1);
  gStyle->SetLineColor(1);
  gStyle->SetPalette(1, 0);
  foo = new TCanvas(); 
  foodal = new TCanvas(); 
  foodal->Size(10, 10);

  hipSetDevice(0);
  runToyFit(argv[1]);

  // Print total minimization time
  double myCPU = stopCPU - startCPU;
  double totalCPU = myCPU; 

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
  std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
  std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
  myCPU = stopProc.tms_utime - startProc.tms_utime;
  std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;

  return 0; 
}
