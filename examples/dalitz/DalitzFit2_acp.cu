// ROOT stuff
#include "TRandom.hh"
#include "TCanvas.h" 
#include "TFile.h" 
#include "TTree.h" 
#include "TH1F.h" 
#include "TH2F.h" 
#include "TStyle.h" 
#include "TRandom3.hh" 
#include "TLegend.h" 
#include "TText.h" 
#include "TLine.h" 

// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

// GooFit stuff
#include "Variable.hh" 
#include "PolynomialPdf.hh" 
#include "DalitzPlotPdf.hh" 
#include "DalitzVetoPdf.hh" 
#include "ResonancePdf.hh" 
#include "AddPdf.hh"
#include "ProdPdf.hh"
#include "GooPdf.hh" 
#include "FitManager.hh" 
#include "UnbinnedDataSet.hh"

using namespace std;

TCanvas* foo; 
TCanvas* foodal; 
timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 
UnbinnedDataSet* data = 0; 

Variable* m12 = 0;
Variable* m13 = 0;
Variable* eventNumber = 0; 
bool fitMasses = false; 
Variable* fixedRhoMass  = new Variable("rho_mass", 0.7758, 0.01, 0.7, 0.8);
Variable* fixedRhoWidth = new Variable("rho_width", 0.1503, 0.01, 0.1, 0.2); 

const fptype _mD0 = 1.86484; 
const fptype _mD02 = _mD0 *_mD0;
const fptype _mD02inv = 1./_mD02; 
const fptype piPlusMass = 0.13957018;
const fptype piZeroMass = 0.1349766;

// Constants used in more than one PDF component. 
Variable* motherM = new Variable("motherM", _mD0);
Variable* chargeM = new Variable("chargeM", piPlusMass);
Variable* neutrlM = new Variable("neutrlM", piZeroMass);
Variable* massSum = new Variable("massSum", _mD0*_mD0 + 2*piPlusMass*piPlusMass + piZeroMass*piZeroMass); // = 3.53481 
Variable* constantOne = new Variable("constantOne", 1); 
Variable* constantZero = new Variable("constantZero", 0); 
  
std::vector<PdfBase*> comps;

GooPdf* kzero_veto = 0; 
char strbuffer[1000]; 
double mesonRad  = 1.5;
DalitzPlotPdf* signalDalitz; 

fptype cpuGetM23 (fptype massPZ, fptype massPM) {
  return (_mD02 + piZeroMass*piZeroMass + piPlusMass*piPlusMass + piPlusMass*piPlusMass - massPZ - massPM); 
}

bool cpuDalitz (fptype m12, fptype m13, fptype bigM, fptype dm1, fptype dm2, fptype dm3) {
  if (m12 < POW(dm1 + dm2, 2)) return false; // This m12 cannot exist, it's less than the square of the (1,2) particle mass.
  if (m12 > POW(bigM - dm3, 2)) return false;   // This doesn't work either, there's no room for an at-rest 3 daughter. 
  
  // Calculate energies of 1 and 3 particles in m12 rest frame. 
  fptype e1star = 0.5 * (m12 - dm2*dm2 + dm1*dm1) / SQRT(m12); 
  fptype e3star = 0.5 * (bigM*bigM - m12 - dm3*dm3) / SQRT(m12); 

  // Bounds for m13 at this value of m12.
  fptype minimum = POW(e1star + e3star, 2) - POW(SQRT(e1star*e1star - dm1*dm1) + SQRT(e3star*e3star - dm3*dm3), 2);
  if (m13 < minimum) return false;
  fptype maximum = POW(e1star + e3star, 2) - POW(SQRT(e1star*e1star - dm1*dm1) - SQRT(e3star*e3star - dm3*dm3), 2);
  if (m13 > maximum) return false;

  return true; 
}

void getToyData (std::string toyFileName) {
  TH2F dalitzplot("dalitzplot", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit); 
  std::vector<Variable*> vars;
  vars.push_back(m12);
  vars.push_back(m13);
  vars.push_back(eventNumber); 
  data = new UnbinnedDataSet(vars); 

  const string suffix = ".root";
  if (toyFileName.rfind(suffix)+suffix.length() == toyFileName.length()){
      TFile*f = TFile::Open(toyFileName.c_str());
      TTree*t = (TTree*)f->Get("ntp");
      assert(t);
      float m2_12, m2_13;
      t->SetBranchAddress("m12", &m2_12);
      t->SetBranchAddress("m13", &m2_13);
      for (int i=0;i<t->GetEntries();i++){
          t->GetEntry(i);
          m12->value = m2_12;
          m13->value = m2_13;
          eventNumber->value = data->getNumEvents(); 
          data->addEvent(); 
          dalitzplot.Fill(m12->value, m13->value); 
      }
      f->Close();
  }
  else{
  std::ifstream reader;
  reader.open(toyFileName.c_str()); 
  std::string buffer;
  while (!reader.eof()) {
    reader >> buffer;
    if (buffer == "====") break; 
    std::cout << buffer; 
  }

  double dummy = 0; 
  while (!reader.eof()) {
    reader >> dummy;
    reader >> dummy;      // m23, m(pi+ pi-), called m12 in processToyRoot convention. 
    reader >> m12->value; // Already swapped according to D* charge. m12 = m(pi+pi0)
    reader >> m13->value;

    // Errors on Dalitz variables
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 

    reader >> dummy; // Decay time
    reader >> dummy; // sigma_t

    reader >> dummy; // Md0
    reader >> dummy; // deltaM
    reader >> dummy; // ProbSig
    reader >> dummy; // Dst charge
    reader >> dummy; // Run
    reader >> dummy; // Event
    reader >> dummy; // Signal and four bkg fractions. 
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 
    reader >> dummy; 

    // EXERCISE 1 (preliminary): Impose an artificial reconstruction efficiency
    // by throwing out events with a probability linear in m12. 
    // NB! This exercise continues below. 

    // EXERCISE 2: Instead of the above efficiency, impose a 
    // K0 veto, by throwing out events with 0.475 < m23 < 0.505. 
/*    double M_23 = cpuGetM23(m12->value, m13->value);
    if (M_23>0.475*0.475 & M_23<0.505*0.505) continue;*/

    // EXERCISE 3: Use both the above. 

    eventNumber->value = data->getNumEvents(); 
    data->addEvent(); 

    dalitzplot.Fill(m12->value, m13->value); 
  }}

  dalitzplot.SetStats(false); 
  dalitzplot.Draw("colz");
  foodal->SaveAs("dalitzplot.png"); 
}

GooPdf* makeKzeroVeto () {
  if (kzero_veto) return kzero_veto; 

  VetoInfo* kVetoInfo = new VetoInfo();
  kVetoInfo->cyclic_index = PAIR_23; 
  kVetoInfo->minimum = new Variable("veto_min", 0.475*0.475);
  kVetoInfo->maximum = new Variable("veto_max", 0.505*0.505);
  vector<VetoInfo*> vetos; vetos.push_back(kVetoInfo); 
  kzero_veto = new DalitzVetoPdf("kzero_veto", m12, m13, motherM, neutrlM, chargeM, chargeM, vetos); 
  return kzero_veto;
}

DalitzPlotPdf* makeSignalPdf (GooPdf* eff = 0) {
  DecayInfo* dtop0pp = new DecayInfo();
  dtop0pp->motherMass  = _mD0; 
  dtop0pp->daug1Mass  = piZeroMass;
  dtop0pp->daug2Mass  = piPlusMass;
  dtop0pp->daug3Mass  = piPlusMass;
  dtop0pp->meson_radius  = 1.5; 
 
  ResonancePdf* rhop  = new ResonancePdf("rhop",
							     new Variable("rhop_amp_real", 1),
							     new Variable("rhop_amp_imag", 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_12);


  bool fixAmps = false;

  ResonancePdf* rhom  = new ResonancePdf("rhom", 
							     fixAmps ? new Variable("rhom_amp_real", 0.714) : 
							     new Variable("rhom_amp_real",  0.714, 0.001, 0, 0),
							     fixAmps ? new Variable("rhom_amp_imag", -0.025) :
							     new Variable("rhom_amp_imag", -0.025, 0.1, 0, 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_13);

  ResonancePdf* rho0  = new ResonancePdf("rho0", 
							     fixAmps ? new Variable("rho0_amp_real", 0.565) : 
							     new Variable("rho0_amp_real", 0.565, 0.001, 0, 0),
							     fixAmps ? new Variable("rho0_amp_imag", 0.164) :
							     new Variable("rho0_amp_imag", 0.164, 0.1, 0, 0),
							     fixedRhoMass,
							     fixedRhoWidth,
							     1,
							     PAIR_23);

  Variable* sharedMass = new Variable("rhop_1450_mass", 1.465, 0.01, 1.0, 2.0);
  Variable* shareWidth = new Variable("rhop_1450_width", 0.400, 0.01, 0.01, 5.0); 

  ResonancePdf* rhop_1450  = new ResonancePdf("rhop_1450", 
								  fixAmps ? new Variable("rhop_1450_amp_real", -0.174) : 
								  new Variable("rhop_1450_amp_real", -0.174, 0.001, 0, 0),
								  fixAmps ? new Variable("rhop_1450_amp_imag", -0.117) :
								  new Variable("rhop_1450_amp_imag", -0.117, 0.1, 0, 0),
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_12);

  ResonancePdf* rho0_1450  = new ResonancePdf("rho0_1450", 
								  fixAmps ? new Variable("rho0_1450_amp_real", 0.325) : 
								  new Variable("rho0_1450_amp_real", 0.325, 0.001, 0, 0),
								  fixAmps ? new Variable("rho0_1450_amp_imag", 0.057) : 
								  new Variable("rho0_1450_amp_imag", 0.057, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_23);

  ResonancePdf* rhom_1450  = new ResonancePdf("rhom_1450", 
								  fixAmps ? new Variable("rhom_1450_amp_real", 0.788) : 
								  new Variable("rhom_1450_amp_real", 0.788, 0.001, 0, 0),
								  fixAmps ? new Variable("rhom_1450_amp_imag", 0.226) : 
								  new Variable("rhom_1450_amp_imag", 0.226, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_13);

  sharedMass = new Variable("rhop_1700_mass",  1.720, 0.01, 1.6, 1.9);
  shareWidth = new Variable("rhop_1700_width", 0.250, 0.01, 0.1, 1.0); 

  
  ResonancePdf* rhop_1700  = new ResonancePdf("rhop_1700", 
								  fixAmps ? new Variable("rhop_1700_amp_real", 2.151) : 
								  new Variable("rhop_1700_amp_real",  2.151, 0.001, 0, 0),
								  fixAmps ? new Variable("rhop_1700_amp_imag", -0.658) : 
								  new Variable("rhop_1700_amp_imag", -0.658, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_12);
  
  ResonancePdf* rho0_1700  = new ResonancePdf("rho0_1700", 
								  fixAmps ? new Variable("rho0_1700_amp_real",  2.400) : 
								  new Variable("rho0_1700_amp_real",  2.400, 0.001, 0, 0),
								  fixAmps ? new Variable("rho0_1700_amp_imag", -0.734) : 
								  new Variable("rho0_1700_amp_imag", -0.734, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_23);
  
  ResonancePdf* rhom_1700  = new ResonancePdf("rhom_1700", 
								  fixAmps ? new Variable("rhom_1700_amp_real",  1.286) : 
								  new Variable("rhom_1700_amp_real",  1.286, 0.001, 0, 0),
								  fixAmps ? new Variable("rhom_1700_amp_imag", -1.532) : 
								  new Variable("rhom_1700_amp_imag", -1.532, 0.1, 0, 0),  
								  sharedMass,
								  shareWidth,
								  1,
								  PAIR_13);
  
  ResonancePdf* f0_980  = new ResonancePdf("f0_980", 
							       fixAmps ? new Variable("f0_980_amp_real",  0.008 * (-_mD02)) : 
							       new Variable("f0_980_amp_real",  0.008 * (-_mD02), 0.001, 0, 0),
							       fixAmps ? new Variable("f0_980_amp_imag", -0.013 * (-_mD02)) : 
							       new Variable("f0_980_amp_imag", -0.013 * (-_mD02), 0.1, 0, 0),  
							       new Variable("f0_980_mass",     0.980, 0.01, 0.8, 1.2),
							       new Variable("f0_980_width",    0.044, 0.001, 0.001, 0.08),
							       (unsigned int)0,
							       PAIR_23);
  
  ResonancePdf* f0_1370  = new ResonancePdf("f0_1370", 
								fixAmps ? new Variable("f0_1370_amp_real", -0.058 * (-_mD02)) : 
								new Variable("f0_1370_amp_real", -0.058 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1370_amp_imag",  0.026 * (-_mD02)) : 
								new Variable("f0_1370_amp_imag",  0.026 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1370_mass",     1.434, 0.01, 1.2, 1.6),
								new Variable("f0_1370_width",    0.173, 0.01, 0.01, 0.4),
								(unsigned int)0,
								PAIR_23);
  
  ResonancePdf* f0_1500  = new ResonancePdf("f0_1500", 
								fixAmps ? new Variable("f0_1500_amp_real", 0.057 * (-_mD02)) : 
								new Variable("f0_1500_amp_real", 0.057 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1500_amp_imag", 0.012 * (-_mD02)) : 
								new Variable("f0_1500_amp_imag", 0.012 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1500_mass",     1.507, 0.01, 1.3, 1.7),
								new Variable("f0_1500_width",    0.109, 0.01, 0.01, 0.3),
								(unsigned int)0,
								PAIR_23);
  
  ResonancePdf* f0_1710  = new ResonancePdf("f0_1710", 
								fixAmps ? new Variable("f0_1710_amp_real", 0.070 * (-_mD02)) : 
								new Variable("f0_1710_amp_real", 0.070 * (-_mD02), 0.001, 0, 0),
								fixAmps ? new Variable("f0_1710_amp_imag", 0.087 * (-_mD02)) : 
								new Variable("f0_1710_amp_imag", 0.087 * (-_mD02), 0.1, 0, 0),  
								new Variable("f0_1710_mass",     1.714, 0.01, 1.5, 2.9), 
								new Variable("f0_1710_width",    0.140, 0.01, 0.01, 0.5),
								(unsigned int)0,
								PAIR_23);
  
  ResonancePdf* f2_1270  = new ResonancePdf("f2_1270", 
								fixAmps ? new Variable("f2_1270_amp_real", -1.027 * (-_mD02inv)) : 
								new Variable("f2_1270_amp_real", -1.027 * (-_mD02inv), 0.001, 0, 0),
								fixAmps ? new Variable("f2_1270_amp_imag", -0.162 * (-_mD02inv)) : 
								new Variable("f2_1270_amp_imag", -0.162 * (-_mD02inv), 0.1, 0, 0),  
								new Variable("f2_1270_mass",     1.2754, 0.01, 1.0, 1.5),
								new Variable("f2_1270_width",    0.1851, 0.01, 0.01, 0.4),
								2,
								PAIR_23);
  
  ResonancePdf* f0_600  = new ResonancePdf("f0_600", 
							       fixAmps ? new Variable("f0_600_amp_real", 0.068 * (-_mD02)) : 
							       new Variable("f0_600_amp_real", 0.068 * (-_mD02), 0.001, 0, 0),
							       fixAmps ? new Variable("f0_600_amp_imag", 0.010 * (-_mD02)) : 
							       new Variable("f0_600_amp_imag", 0.010 * (-_mD02), 0.1, 0, 0),  
							       new Variable("f0_600_mass",     0.500, 0.01, 0.3, 0.7),
							       new Variable("f0_600_width",    0.400, 0.01, 0.2, 0.6), 
							       (unsigned int)0,
							       PAIR_23);
  
  ResonancePdf* nonr  = new ResonancePdf("nonr",
							     fixAmps ? new Variable("nonr_amp_real", 0.5595 * (-1)) : 
							     new Variable("nonr_amp_real", 0.5595 * (-1),   0.001, 0, 0),
							     fixAmps ? new Variable("nonr_amp_imag", -0.108761 * (-1)) : 
							     new Variable("nonr_amp_imag", -0.108761* (-1), 0.1, 0, 0)); 

  dtop0pp->resonances.push_back(nonr); 
  dtop0pp->resonances.push_back(rhop);
  dtop0pp->resonances.push_back(rho0); 
  dtop0pp->resonances.push_back(rhom); 
  dtop0pp->resonances.push_back(rhop_1450); 
  dtop0pp->resonances.push_back(rho0_1450); 
  dtop0pp->resonances.push_back(rhom_1450); 
  dtop0pp->resonances.push_back(rhop_1700); 
  dtop0pp->resonances.push_back(rho0_1700); 
  dtop0pp->resonances.push_back(rhom_1700); 
  dtop0pp->resonances.push_back(f0_980); 
  dtop0pp->resonances.push_back(f0_1370); 
  dtop0pp->resonances.push_back(f0_1500); 
  dtop0pp->resonances.push_back(f0_1710); 
  dtop0pp->resonances.push_back(f2_1270); 
  dtop0pp->resonances.push_back(f0_600); 

  if (!fitMasses) {
    for (vector<ResonancePdf*>::iterator res = dtop0pp->resonances.begin(); res != dtop0pp->resonances.end(); ++res) {
      (*res)->setParameterConstantness(true); 
    }
  }

  if (!eff) {
    // By default create a constant efficiency. 
    vector<Variable*> offsets;
    vector<Variable*> observables;
    vector<Variable*> coefficients; 

    observables.push_back(m12);
    observables.push_back(m13);
    offsets.push_back(constantZero);
    offsets.push_back(constantZero);
    coefficients.push_back(constantOne); 
    eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
  }
  comps.clear();
  comps.push_back(eff);
  if (!kzero_veto) makeKzeroVeto();
  comps.push_back(kzero_veto);
  ProdPdf* effWithVeto = new ProdPdf("effWithVeto", comps);

  return new DalitzPlotPdf("signalPDF", m12, m13, eventNumber, dtop0pp, effWithVeto);
}

void drawFitPlotsWithPulls(TH1* hd, TH1* ht, string plotdir){
    const char* hname = hd->GetName();
    char obsname[10];
    for (int i=0;;i++) {
        if (hname[i]=='_') obsname[i] = '\0';
        else obsname[i] = hname[i];
        if (obsname[i] == '\0') break;
    }
    ht->Scale(hd->Integral()/ht->Integral());
    foo->cd(); 
    foo->Clear();
    hd->Draw("ep");
    ht->Draw("csame");
    sprintf(strbuffer, "%s/%s_fit.C", plotdir.c_str(), obsname);
    foo->SaveAs(strbuffer);
    sprintf(strbuffer, "%s/%s_fit.pdf", plotdir.c_str(), obsname);
    foo->SaveAs(strbuffer);
/*    sprintf(strbuffer, "%s/%s_fit_log.pdf", plotdir.c_str(), obsname);
    foo->SaveAs(strbuffer);*/
}

void makeToyDalitzPdfPlots (GooPdf* overallSignal, string plotdir = "plots") {
  TH1F m12_dat_hist("m12_dat_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit);
  m12_dat_hist.SetStats(false); 
  m12_dat_hist.SetMarkerStyle(8); 
  m12_dat_hist.SetMarkerSize(1.2);
  m12_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV]");
  sprintf(strbuffer, "Events / %.1f MeV", 1e3*m12_dat_hist.GetBinWidth(1));
  m12_dat_hist.GetYaxis()->SetTitle(strbuffer); 
  TH1F m12_pdf_hist("m12_pdf_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit);
  m12_pdf_hist.SetStats(false); 
  m12_pdf_hist.SetLineColor(kBlue); 
  m12_pdf_hist.SetLineWidth(3); 
  TH1F m13_dat_hist("m13_dat_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m13_dat_hist.SetStats(false); 
  m13_dat_hist.SetMarkerStyle(8); 
  m13_dat_hist.SetMarkerSize(1.2);
  m13_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV]");
  sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
  m13_dat_hist.GetYaxis()->SetTitle(strbuffer); 
  TH1F m13_pdf_hist("m13_pdf_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m13_pdf_hist.SetStats(false); 
  m13_pdf_hist.SetLineColor(kBlue); 
  m13_pdf_hist.SetLineWidth(3); 
  TH1F m23_dat_hist("m23_dat_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m23_dat_hist.SetStats(false); 
  m23_dat_hist.SetMarkerStyle(8); 
  m23_dat_hist.SetMarkerSize(1.2);
  m23_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{-}) [GeV]");
  sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
  m23_dat_hist.GetYaxis()->SetTitle(strbuffer); 
  TH1F m23_pdf_hist("m23_pdf_hist", "", m13->numbins, m13->lowerlimit, m13->upperlimit);
  m23_pdf_hist.SetStats(false); 
  m23_pdf_hist.SetLineColor(kBlue); 
  m23_pdf_hist.SetLineWidth(3); 
  double totalPdf = 0; 
  double totalDat = 0; 
  TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit);
  dalitzpp0_dat_hist.SetStats(false); 
  dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV]");
  dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV]");
  TH2F dalitzpp0_pdf_hist("dalitzpp0_pdf_hist", "", m12->numbins, m12->lowerlimit, m12->upperlimit, m13->numbins, m13->lowerlimit, m13->upperlimit);
/*  dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(K^{-} #pi^{0}) [GeV^{2}]");
  dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(K^{-} #pi^{+}) [GeV^{2}]");*/
  dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV^{2}]");
  dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV^{2}]");
  dalitzpp0_pdf_hist.SetStats(false); 
    std::vector<Variable*> vars;
    vars.push_back(m12);
    vars.push_back(m13);
    vars.push_back(eventNumber); 
    UnbinnedDataSet currData(vars); 
  int evtCounter = 0; 

  for (int i = 0; i < m12->numbins; ++i) {
      m12->value = m12->lowerlimit + (m12->upperlimit - m12->lowerlimit)*(i + 0.5) / m12->numbins; 
      for (int j = 0; j < m13->numbins; ++j) {
          m13->value = m13->lowerlimit + (m13->upperlimit - m13->lowerlimit)*(j + 0.5) / m13->numbins; 
          if (!cpuDalitz(m12->value, m13->value, _mD0, piPlusMass, piPlusMass, piZeroMass)) continue;
          eventNumber->value = evtCounter; 
          evtCounter++;
          currData.addEvent(); 
      }
  }
  overallSignal->setData(&currData);
  signalDalitz->setDataSize(currData.getNumEvents()); 
  std::vector<std::vector<double> > pdfValues;
  overallSignal->getCompProbsAtDataPoints(pdfValues);
  for (unsigned int j = 0; j < pdfValues[0].size(); ++j) {
	double currm12 = currData.getValue(m12, j);
	double currm13 = currData.getValue(m13, j);

      dalitzpp0_pdf_hist.Fill(currm12, currm13, pdfValues[0][j]);
      m12_pdf_hist.Fill(currm12, pdfValues[0][j]);
      m13_pdf_hist.Fill(currm13, pdfValues[0][j]);
      m23_pdf_hist.Fill(cpuGetM23(currm12, currm13), pdfValues[0][j]); 
      totalPdf     += pdfValues[0][j]; 
  }
  foodal->cd(); 
  foodal->SetLogz(false);
  dalitzpp0_pdf_hist.Draw("colz");
  foodal->SaveAs((plotdir + "/dalitzpp0_pdf.png").c_str());
/*  m12_pdf_hist.Draw("");
  foodal->SaveAs((plotdir + "/m12_pdf_hist.png").c_str());
  m13_pdf_hist.Draw("");
  foodal->SaveAs((plotdir + "/m13_pdf_hist.png").c_str());
  if (!data) return;*/
  for (unsigned int evt = 0; evt < data->getNumEvents(); ++evt) {
    double data_m12 = data->getValue(m12, evt);
    m12_dat_hist.Fill(data_m12); 
    double data_m13 = data->getValue(m13, evt);
    m13_dat_hist.Fill(data_m13); 
    dalitzpp0_dat_hist.Fill(data_m12, data_m13);
    m23_dat_hist.Fill(cpuGetM23(data_m12, data_m13)); 
    totalDat++; 
  }
  dalitzpp0_dat_hist.Draw("colz");
  foodal->SaveAs((plotdir + "/dalitzpp0_dat.png").c_str());

  drawFitPlotsWithPulls(&m12_dat_hist, &m12_pdf_hist, plotdir);
  drawFitPlotsWithPulls(&m13_dat_hist, &m13_pdf_hist, plotdir);
  drawFitPlotsWithPulls(&m23_dat_hist, &m23_pdf_hist, plotdir);
}

void runToyFit (std::string toyFileName) {
  m12 = new Variable("m12", 0, 3);
  m13 = new Variable("m13", 0, 3); 
  m12->numbins = 1000;
  m13->numbins = 1000;
  eventNumber = new Variable("eventNumber", 0, INT_MAX);
  getToyData(toyFileName);

  // EXERCISE 1 (real part): Create a PolynomialPdf which models
  // the efficiency you imposed in the preliminary, and use it in constructing
  // the signal PDF. 

  // EXERCISE 2: Create a K0 veto function and use it as the efficiency. 

  // EXERCISE 3: Make the efficiency a product of the two functions
  // from the previous exercises.

  signalDalitz = makeSignalPdf(); 
  comps.clear();
  comps.push_back(signalDalitz);
  ProdPdf* overallSignal = new ProdPdf("overallSignal", comps);
  overallSignal->setData(data); 
  signalDalitz->setDataSize(data->getNumEvents()); 
  FitManager datapdf(overallSignal); 
  
  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);
  datapdf.fit(); 
  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  makeToyDalitzPdfPlots(overallSignal);   
}

int main (int argc, char** argv) {
  gStyle->SetCanvasBorderMode(0);
  gStyle->SetCanvasColor(10);
  gStyle->SetFrameFillColor(10);
  gStyle->SetFrameBorderMode(0);
  gStyle->SetPadColor(0);
  gStyle->SetTitleColor(1);
  gStyle->SetStatColor(0);
  gStyle->SetFillColor(0);
  gStyle->SetFuncWidth(1);
  gStyle->SetLineWidth(1);
  gStyle->SetLineColor(1);
  gStyle->SetPalette(1, 0);
  foo = new TCanvas(); 
  foodal = new TCanvas(); 
  foodal->Size(10, 10);

  hipSetDevice(0);
  runToyFit(argv[1]);

  // Print total minimization time
  double myCPU = stopCPU - startCPU;
  double totalCPU = myCPU; 

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
  std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
  std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
  myCPU = stopProc.tms_utime - startProc.tms_utime;
  std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;

  return 0; 
}
